#include "hip/hip_runtime.h"
#include <SLOptixRaytracerHelper.h>
#include <SLOptixDefinitions.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __miss__radiance()
{
    auto* rt_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    setColor(rt_data->bg_color);
}

extern "C" __global__ void __miss__occlusion()
{
}

extern "C" __global__ void __anyhit__radiance()
{
}

extern "C" __global__ void __anyhit__occlusion()
{
    auto *rt_data = reinterpret_cast<HitData *>( optixGetSbtDataPointer());
    if (length(rt_data->material.emissive_color) > 0.0f) {
        // If the hit material is emissive set this as the light color
        setColor(rt_data->material.emissive_color);
    } else {
        // Add the kt value of the hit material to the occlusion value
        float occlusion = getOcclusion() + (1.0f - rt_data->material.kt);
        setOcclusion(occlusion);
        // If the occlusion reaches one then we can terminate the ray
        if(occlusion >= 1) {
            optixTerminateRay();
        }
    }
}

extern "C" __global__ void __closesthit__radiance() {
    // Get all data for the hit point
    auto *rt_data = reinterpret_cast<HitData *>( optixGetSbtDataPointer());
    unsigned int idx = optixGetPrimitiveIndex();
    const float3 ray_dir = optixGetWorldRayDirection();
    float4 color = make_float4(0.0f);

    // calculate normal vector
    float3 N;
    {
        const float2 barycentricCoordinates = optixGetTriangleBarycentrics();
        const float u = barycentricCoordinates.x;
        const float v = barycentricCoordinates.y;
        if (rt_data->normals && rt_data->indices) {
            // Interpolate normal vector with barycentric coordinates
            N = (1.f-u-v) * rt_data->normals[rt_data->indices[idx].x]
                +         u * rt_data->normals[rt_data->indices[idx].y]
                +         v * rt_data->normals[rt_data->indices[idx].z];
            N = normalize( optixTransformNormalFromObjectToWorldSpace( N ) );
        } else {
            OptixTraversableHandle gas = optixGetGASTraversableHandle();
            float3 vertex[3] = { make_float3(0.0f), make_float3(0.0f), make_float3(0.0f)};
            optixGetTriangleVertexData(gas,
                                       idx,
                                       rt_data->sbtIndex,
                                       0,
                                       vertex);
            N = normalize(cross(vertex[1] - vertex[0], vertex[2] - vertex[0]));
        }

        if (rt_data->textureObject) {
            const float2 tc
                    = (1.f-u-v) * rt_data->texCords[rt_data->indices[idx].x]
                      +         u * rt_data->texCords[rt_data->indices[idx].y]
                      +         v * rt_data->texCords[rt_data->indices[idx].z];
            color = tex2D<float4>(rt_data->textureObject, tc.x, tc.y);

            const uint3 idx = optixGetLaunchIndex();
            params.debug[idx.y * params.width + idx.x] = make_float3(tc.x, tc.y, 0.0f);
        }
    }

    // if a back face was hit then the normal vector is in the opposite direction
    if (optixIsTriangleBackFaceHit()) {
        N = N * -1;
    }
    // calculate hit point
    const float3 P = optixGetWorldRayOrigin() + optixGetRayTmax() * ray_dir;

    // calculate local illumination for every light source
    for (int i = 0; i < params.numLights; i++) {
        const float Ldist = length(params.lights[i].position - P);
        const float3 L = normalize(params.lights[i].position - P);
        const float nDl = dot(L, N);

        // Phong specular reflection
//        const float3 R = normalize(reflect(-L, N));
//        const float3 V = normalize(-ray_dir);
//        powf( max(dot(R, V), 0.0), rt_data->material.shininess )
        // Blinn specular reflection
        const float3 H = normalize(L - ray_dir); // half vector between light & eye

        uint32_t p0 = float_as_int( 1.0f );
        uint32_t p1 = float_as_int( 1.0f );
        uint32_t p2 = float_as_int( 1.0f );
        uint32_t p3 = float_as_int( 1.0f );
        uint32_t p4 = float_as_int( 0.0f );
        if ( nDl > 0.0f)
        {
            // Send shadow ray
            optixTrace(
                    params.handle,
                    P,
                    L,
                    1e-3f,                         // tmin
                    Ldist,                               // tmax
                    0.0f,                       // rayTime
                    OptixVisibilityMask( 1 ),
                    OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT | OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
                    RAY_TYPE_OCCLUSION,        // SBT offset
                    RAY_TYPE_COUNT,            // SBT stride
                    RAY_TYPE_OCCLUSION,     // missSBTIndex
                    p0, p1, p2, p3, p4 // payload
            );
        }

        float4 light_color = make_float4(int_as_float( p0 ), int_as_float( p1 ), int_as_float( p2 ), int_as_float( p3 ));
        float occlusion = int_as_float( p4 );
        occlusion = min(occlusion, 1.0f);

        // Phong shading
        if (occlusion < 1) {
//            color += (rt_data->material.specular_color * powf( max(dot(N, H), 0.0), rt_data->material.shininess )   // specular
//                     + rt_data->material.diffuse_color * max(nDl, 0.0f))                                            // diffuse
//                    * (1.0f - occlusion)                                                                            // occlusion
//                    * light_color                                                                                   // multiply with light color
//                    * lightAttenuation(params.lights[i], Ldist);                                                    // multiply with light attenuation
        }
//        color += rt_data->material.ambient_color * lightAttenuation(params.lights[i], Ldist);
    }

    // Send reflection ray
    if(getDepth() < params.max_depth && rt_data->material.kr > 0.0f) {
//        color += (traceRadianceRay(params.handle, P, reflect(ray_dir, N), getRefractionIndex(), getDepth() + 1) * rt_data->material.kr);
    }

    // The color value so far is only as strong as the light that does not pass through the object
//    color *= (1.0f - rt_data->material.kt);

    // Send refraction ray
    if(getDepth() < params.max_depth && rt_data->material.kt > 0.0f) {
        // calculate eta
        float refractionIndex = rt_data->material.kn;
        float eta = getRefractionIndex() / rt_data->material.kn;
        if (optixIsTriangleBackFaceHit()) {
            refractionIndex = 1.0f;
            eta = rt_data->material.kn / 1.0f;
        }

        // calculate transmission vector T
        float3 T;
        float c1 = dot(N, -ray_dir);
        float w = eta * c1;
        float c2 = 1.0f + (w - eta) * (w + eta);
        if(c2 >= 0.0f) {
            T = eta * ray_dir + (w - sqrtf(c2)) * N;
        } else {
            T = 2.0f * (dot(-ray_dir, N)) * N  + ray_dir;
        }
//        color += (traceRadianceRay(params.handle, P, T, refractionIndex, getDepth() + 1) * rt_data->material.kt);
    }

    // Add emissive and ambient to current color
//    color += rt_data->material.emissive_color;
//    color += (rt_data->material.ambient_color * params.globalAmbientColor);

    // Set color to payload
    setColor(color);
}