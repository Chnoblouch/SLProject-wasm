#include "hip/hip_runtime.h"
#include <SLOptixHelper.h>
#include <SLOptixDefinitions.h>
#include <hip/hip_runtime_api.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __miss__sample() {
    auto *rt_data = reinterpret_cast<MissData *>( optixGetSbtDataPointer());
    setColor(rt_data->bg_color);
}