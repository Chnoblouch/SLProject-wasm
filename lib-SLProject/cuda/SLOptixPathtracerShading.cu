#include "hip/hip_runtime.h"
#include <SLOptixHelper.h>
#include <SLOptixDefinitions.h>
#include <hip/hip_runtime_api.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __miss__sample() {
    auto *rt_data = reinterpret_cast<MissData *>( optixGetSbtDataPointer());
    setColor(rt_data->bg_color);
}

extern "C" __global__ void __anyhit__radiance() {
}

static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
    // Uniformly sample disk.
    const float r   = u1;
    const float phi = 2.0f*M_PIf * u2;
    p.x = r * cosf( phi );
    p.y = r * sinf( phi );

    // Project up to hemisphere.
    p.z = sqrtf( fmaxf( 0.0f, 1.0f - p.x*p.x - p.y*p.y ) );
}

extern "C" __global__ void __closesthit__radiance() {
    // Get all data for the hit point
    auto *rt_data = reinterpret_cast<HitData *>( optixGetSbtDataPointer());
    const float3 ray_dir = optixGetWorldRayDirection();

    uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    hiprandState *state = &params.states[idx.y * dim.x + idx.x];

    // calculate normal vector
    float3 N = getNormalVector();
    // calculate texture color
    float4 texture_color = getTextureColor();

    // calculate hit point
    const float3 P = optixGetWorldRayOrigin() + optixGetRayTmax() * ray_dir;

    if (rt_data->material.emissive_color.x != 0 ||
        rt_data->material.emissive_color.y != 0 ||
        rt_data->material.emissive_color.z != 0) {
        setColor(rt_data->material.emissive_color);
    } else {
        // initialize color
        float4 local_color;
        float4 incoming_color;
        if (getDepth() < params.max_depth) {
            if (rt_data->material.kr > hiprand_uniform(state)) {
                incoming_color = traceReflectionRay(params.handle, P, N, ray_dir);
                local_color = rt_data->material.specular_color;
            } else if (rt_data->material.kt > hiprand_uniform(state)) {
                incoming_color = traceRefractionRay(params.handle, P, N, ray_dir, rt_data->material.kn);
                local_color = rt_data->material.transmissiv_color;
            } else {
//                float3 direction = make_float3((hiprand_uniform(state) * 2) - 1.0f, (hiprand_uniform(state) * 2) - 1.0f, (hiprand_uniform(state) * 2) - 1.0f);
                float3 direction;
                cosine_sample_hemisphere( hiprand_uniform(state), hiprand_uniform(state), direction );
                incoming_color = traceSecondaryRay(params.handle, P, direction);
                local_color = rt_data->material.diffuse_color;
            }
            local_color *= texture_color;
        }

        // Set color to payload
        setColor(local_color * incoming_color);
    }
}