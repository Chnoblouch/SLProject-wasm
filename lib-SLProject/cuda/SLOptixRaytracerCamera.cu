#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optix_types.h>

#include <hip/hip_runtime.h>
#include <SLOptixDefinitions.h>

extern "C" {
__constant__ Params params;
}

extern "C"
__global__ void __raygen__draw_solid_color()
{
    optixThrowException(1);
    uint3 launch_index = optixGetLaunchIndex();
    CameraData* rtData = (CameraData*)optixGetSbtDataPointer();
    params.image[launch_index.y * params.image_width + launch_index.x] = make_uchar3( 255, 0, 0 );
}